#include "hip/hip_runtime.h"
#include "gelu.h"
#include <math.h>

namespace Tn {
    template<typename T>
    void write(char*& buffer, const T& val) {
        *reinterpret_cast<T*>(buffer) = val;
        buffer += sizeof(T);
    }

    template<typename T>
    void read(const char*& buffer, T& val) {
        val = *reinterpret_cast<const T*>(buffer);
        buffer += sizeof(T);
    }
}  

namespace nvinfer1
{
    gelu::gelu()
    {
    }

    gelu::~gelu()
    {
    }
    // create the plugin at runtime from a byte stream
    gelu::gelu(const void* data, size_t length)
    {
        const char *d = reinterpret_cast<const char *>(data), *a = d;
        Tn::read(d, mInputSize);
        assert(d == a + length);
    }

    void gelu::serialize(void* buffer) const noexcept
    {
        char* d = static_cast<char*>(buffer), *a = d;
        Tn::write(d, mInputSize);
        assert(d == a + getSerializationSize());
    }

    size_t gelu::getSerializationSize() const noexcept
    {
        return sizeof(mInputSize);
    }

    int gelu::initialize() noexcept
    {
        return 0;
    }

    Dims gelu::getOutputDimensions(int index, const Dims* inputs, int nbInputDims)  noexcept
    {
        assert(nbInputDims == 1);
        Dims outputDims;
        outputDims.nbDims = inputs[0].nbDims;
        for (int i = 0; i < inputs[0].nbDims; i++) {
            outputDims.d[i] = inputs[0].d[i];
        }
        return outputDims;
    }

    // Set plugin namespace
    void gelu::setPluginNamespace(const char* pluginNamespace) noexcept
    {
        mPluginNamespace = pluginNamespace;
    }

    const char* gelu::getPluginNamespace() const noexcept
    {
        return mPluginNamespace;
    }

    // Return the DataType of the plugin output at the requested index
    DataType gelu::getOutputDataType(int index, const nvinfer1::DataType* inputTypes, int nbInputs) const  noexcept
    {
        return DataType::kFLOAT;
    }

    // Return true if output tensor is broadcast across a batch.
    bool gelu::isOutputBroadcastAcrossBatch(int outputIndex, const bool* inputIsBroadcasted, int nbInputs) const noexcept
    {
        return false;
    }

    // Return true if plugin can use input that is broadcast across batch without replication.
    bool gelu::canBroadcastInputAcrossBatch(int inputIndex) const noexcept
    {
        return false;
    }

    void gelu::configurePlugin(const PluginTensorDesc* in, int nbInput, const PluginTensorDesc* out, int nbOutput) noexcept
    {

        mInputSize = 1;
        for (int i = 0; i < in[0].dims.nbDims; i++) {
            mInputSize *= in[0].dims.d[i];
        }
    }

    // Attach the plugin object to an execution context and grant the plugin the access to some context resource.
    void gelu::attachToContext(cudnnContext* cudnnContext, cublasContext* cublasContext, IGpuAllocator* gpuAllocator) noexcept
    {
    }

    // Detach the plugin object from its execution context.
    void gelu::detachFromContext() noexcept {}

    const char* gelu::getPluginType() const noexcept
    {
        return "geluLayer_TRT";
    }

    const char* gelu::getPluginVersion() const noexcept
    {
        return "1";
    }

    void gelu::destroy() noexcept
    {
        delete this;
    }

    // Clone the plugin
    IPluginV2IOExt* gelu::clone() const noexcept
    {
        gelu *p = new gelu();
        p->setPluginNamespace(mPluginNamespace);
        p->setInputSize(mInputSize);
        return p;
    }

    __global__ void geluKer(const float *in, float *out, int size) {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (idx >= size)
            return;
        // x * 0.5 * (1.0 + torch.erf(x / math.sqrt(2.0)))
        out[idx] = in[idx] * 0.5 *(1.0 + erf(in[idx]/1.4142135381698608));
        // out[idx] = 77;
    }
    void gelu::forwardGpu(const float *const * inputs, float* output, hipStream_t stream, int batchSize) {

        int numElem = batchSize * mInputSize;
        geluKer<<<(numElem + mThreadCount - 1) / mThreadCount, mThreadCount>>>
            (inputs[0], output, numElem);
    }

    int gelu::enqueue(int batchSize, void const * const * inputs, void* const * outputs, void* workspace, hipStream_t stream) noexcept
    {
        forwardGpu((const float *const *)inputs, (float*)outputs[0], stream, batchSize);
        return 0;
    }

    PluginFieldCollection geluCreator::mFC{};
    std::vector<PluginField> geluCreator::mPluginAttributes;

    geluCreator::geluCreator()
    {
        mPluginAttributes.clear();
        mFC.nbFields = mPluginAttributes.size();
        mFC.fields = mPluginAttributes.data();
    }

    const char* geluCreator::getPluginName() const noexcept
    {
            return "geluLayer_TRT";
    }

    const char* geluCreator::getPluginVersion() const noexcept
    {
            return "1";
    }

    const PluginFieldCollection* geluCreator::getFieldNames() noexcept
    {
            return &mFC;
    }

    IPluginV2IOExt* geluCreator::createPlugin(const char* name, const PluginFieldCollection* fc) noexcept
    {
        gelu* obj = new gelu();
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

    IPluginV2IOExt* geluCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength) noexcept
    {
        // This object will be deleted when the network is destroyed, which will
        gelu* obj = new gelu(serialData, serialLength);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }


}

